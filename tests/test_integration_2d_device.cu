/* Copyright (c) 2016 - 2021, the adamantine authors.
 *
 * This file is subject to the Modified BSD License and may not be distributed
 * without copyright and license information. Please refer to the file LICENSE
 * for the text and further information on this license.
 */

#define BOOST_TEST_MODULE Integration_2D_Device

#include "../application/adamantine.hh"

#include <fstream>

#include "main.cc"

BOOST_AUTO_TEST_CASE(intregation_2D_device)
{
  MPI_Comm communicator = MPI_COMM_WORLD;

  std::vector<adamantine::Timer> timers;
  initialize_timers(communicator, timers);

  // Read the input.
  std::string const filename = "integration_2d.info";
  adamantine::ASSERT_THROW(boost::filesystem::exists(filename) == true,
                           "The file " + filename + " does not exist.");
  boost::property_tree::ptree database;
  boost::property_tree::info_parser::read_info(filename, database);

  auto result =
      run<2, dealii::MemorySpace::CUDA>(communicator, database, timers);

  dealii::LinearAlgebra::distributed::Vector<double, dealii::MemorySpace::Host>
      result_host(result.get_partitioner());
  result_host.import(result, dealii::VectorOperation::insert);
  std::ifstream gold_file("integration_2d_gold.txt");
  double const tolerance = 0.1;
  for (unsigned int i = 0; i < result_host.local_size(); ++i)
  {
    double gold_value = -1.;
    gold_file >> gold_value;
    BOOST_CHECK_CLOSE(result_host.local_element(i), gold_value, tolerance);
  }
}
